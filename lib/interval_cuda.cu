#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "radixsort.h"
#include "interval.h"
#include "timer.h"
#include "interval_cuda.h"
#include "reduce.h"
#include "bsearch_cuda.h"
#include "bsearch_cuda.cu"

//{{{ per_interval_count_intersections_bsearch_seq 
void per_interval_count_intersections_bsearch_cuda(struct interval *A,
												  unsigned int size_A,
												  struct interval *B,
												  unsigned int size_B,
												  unsigned int *R)
{
	int block_size = 256;
	dim3 dimBlock(block_size);
	int grid_size = ( size_A + block_size - 1) / (block_size * 1);
	dim3 dimGridSearch( grid_size );
	hipError_t err;

	start(); //data_prep_time

	unsigned int *A_starts_h, *A_lens_h, *B_starts_h, *B_ends_h;
	unsigned int *A_starts_d, *A_lens_d, *B_starts_d, *B_ends_d;
	unsigned int *R_d;
	allocate_and_move(A,
					&A_starts_h,
					&A_starts_d,
					&A_lens_h ,
					&A_lens_d,
					size_A,
					B,
					&B_starts_h ,
					&B_starts_d,
					&B_ends_h ,
					&B_ends_d,
					size_B,
					&R_d);

	stop(); //data_prep_time
	unsigned long data_prep_time = report();

	start(); //sort_time

	//{{{ Sort B_starts and B_ends
	// Sort B by start
	nvRadixSort::RadixSort radixsortB_starts(size_B, true);
	radixsortB_starts.sort((unsigned int*)B_starts_d, 0, size_B, 32);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_starts: %s.\n", hipGetErrorString( err) );

	// Sort B by end
	nvRadixSort::RadixSort radixsortB_ends(size_B, true);
	radixsortB_ends.sort((unsigned int*)B_ends_d, 0, size_B, 32);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_ends: %s.\n", hipGetErrorString( err) );
	//}}}

	stop(); //sort_time
	unsigned long sort_time = report();

	start(); //intersect_time

	//{{{ Compute and count intersections
	count_bsearch_cuda <<<dimGridSearch, dimBlock >>> (
			A_starts_d, A_lens_d, size_A,
			B_starts_d, B_ends_d, size_B,
			R_d,
			1);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_ends: %s.\n", hipGetErrorString( err) );

	hipMemcpy(R, R_d, size_A* sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );

	//}}}
	
	stop(); //intersect_time
	unsigned long intersect_time = report();

	unsigned long total_time = data_prep_time + 
							   sort_time +
							   intersect_time;
	printf("bsearch\t"
		   "total:%lu\t"
		   "prep:%lu,%f\t"
		   "sort:%lu,%f\t"
		   "intersect:%lu,%f\n",
		   total_time,
		   data_prep_time,  (double)data_prep_time / (double)total_time,
		   sort_time, (double)sort_time / (double)total_time,
		   intersect_time, (double)intersect_time / (double)total_time);

	hipFree(A_starts_d);
	hipFree(A_lens_d);
	hipFree(B_starts_d);
	hipFree(B_ends_d);
	hipFree(R_d);
}
//}}}

//{{{ unsigned int count_intersections_bsearch_cuda(struct interval *A,
unsigned int count_intersections_bsearch_cuda(struct interval *A,
										      unsigned int size_A,
											  struct interval *B,
										      unsigned int size_B)
{
	int block_size = 256;
	dim3 dimBlock(block_size);
	int grid_size = ( size_A + block_size - 1) / (block_size * 1);
	dim3 dimGridSearch( grid_size );
	hipError_t err;

	start(); //data_prep_time

	unsigned int *A_starts_h, *A_lens_h, *B_starts_h, *B_ends_h;
	unsigned int *A_starts_d, *A_lens_d, *B_starts_d, *B_ends_d;
	unsigned int *R_d;
	allocate_and_move(A,
					&A_starts_h,
					&A_starts_d,
					&A_lens_h ,
					&A_lens_d,
					size_A,
					B,
					&B_starts_h ,
					&B_starts_d,
					&B_ends_h ,
					&B_ends_d,
					size_B,
					&R_d);

	stop(); //data_prep_time
	unsigned long data_prep_time = report();

	start(); //sort_time

	//{{{ Sort B_starts and B_ends
	// Sort B by start
	nvRadixSort::RadixSort radixsortB_starts(size_B, true);
	radixsortB_starts.sort((unsigned int*)B_starts_d, 0, size_B, 32);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_starts: %s.\n", hipGetErrorString( err) );

	// Sort B by end
	nvRadixSort::RadixSort radixsortB_ends(size_B, true);
	radixsortB_ends.sort((unsigned int*)B_ends_d, 0, size_B, 32);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_ends: %s.\n", hipGetErrorString( err) );
	//}}}

	stop(); //sort_time
	unsigned long sort_time = report();

	start(); //intersect_time

	//{{{ Compute and count intersections
	count_bsearch_cuda <<<dimGridSearch, dimBlock >>> (
			A_starts_d, A_lens_d, size_A,
			B_starts_d, B_ends_d, size_B,
			R_d,
			1);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_ends: %s.\n", hipGetErrorString( err) );

	parallel_sum(R_d, block_size, size_A, 1024);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Parallel sum: %s.\n", hipGetErrorString( err) );


	unsigned int R;
	hipMemcpy(&R, R_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );

	//}}}
	
	stop(); //intersect_time
	unsigned long intersect_time = report();

	unsigned long total_time = data_prep_time + 
							   sort_time +
							   intersect_time;
	printf("bsearch\t"
		   "total:%lu\t"
		   "prep:%lu,%f\t"
		   "sort:%lu,%f\t"
		   "intersect:%lu,%f\n",
		   total_time,
		   data_prep_time,  (double)data_prep_time / (double)total_time,
		   sort_time, (double)sort_time / (double)total_time,
		   intersect_time, (double)intersect_time / (double)total_time);

	hipFree(A_starts_d);
	hipFree(A_lens_d);
	hipFree(B_starts_d);
	hipFree(B_ends_d);
	hipFree(R_d);

	return R;
}
//}}}

//{{{ unsigned int count_intersections_sort_bsearch_cuda(struct interval *A,
unsigned int count_intersections_sort_bsearch_cuda(struct interval *A,
										      unsigned int size_A,
											  struct interval *B,
										      unsigned int size_B)
{
	int block_size = 256;
	dim3 dimBlock(block_size);
	int grid_size = ( size_A + block_size - 1) / (block_size * 1);
	dim3 dimGridSearch( grid_size );
	hipError_t err;

	start(); //data_prep_time
	//{{{ Allocate and move 
	unsigned int *A_starts_h, *A_lens_h, *B_starts_h, *B_ends_h;
	unsigned int *A_starts_d, *A_lens_d, *B_starts_d, *B_ends_d;
	unsigned int *R_d;
	allocate_and_move(A,
					&A_starts_h,
					&A_starts_d,
					&A_lens_h ,
					&A_lens_d,
					size_A,
					B,
					&B_starts_h ,
					&B_starts_d,
					&B_ends_h ,
					&B_ends_d,
					size_B,
					&R_d);
	//}}}
	stop(); //data_prep_time
	unsigned long data_prep_time = report();

	start(); //sort_time
	//{{{ Sort B_starts and B_ends
	// Sort B by start
	nvRadixSort::RadixSort radixsortB_starts(size_B, true);
	radixsortB_starts.sort((unsigned int*)B_starts_d, 0, size_B, 32);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_starts: %s.\n", hipGetErrorString( err) );

	// Sort B by end
	nvRadixSort::RadixSort radixsortB_ends(size_B, true);
	radixsortB_ends.sort((unsigned int*)B_ends_d, 0, size_B, 32);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_ends: %s.\n", hipGetErrorString( err) );
	//}}}
	stop();	//sort_time
	unsigned long sort_time = report();
	
	start();
	//{{{ Sort A
	nvRadixSort::RadixSort sort_A_starts_lens_d(size_A, false);
	sort_A_starts_lens_d.sort((unsigned int*)A_starts_d, A_lens_d, size_A, 32);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort A_starts and lens: %s.\n", hipGetErrorString( err) );

	stop();
	unsigned long sort_q_time = report();
	//}}}
	stop();	//pre_sort_time
	unsigned long pre_sort_time = report();

	start();
	//{{{ Compute and count intersections
	count_bsearch_cuda <<<dimGridSearch, dimBlock >>> (
			A_starts_d, A_lens_d, size_A,
			B_starts_d, B_ends_d, size_B,
			R_d,
			1);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_ends: %s.\n", hipGetErrorString( err) );

	parallel_sum(R_d, block_size, size_A, 1024);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Parallel sum: %s.\n", hipGetErrorString( err) );


	unsigned int R;
	hipMemcpy(&R, R_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );

	//}}}
	stop(); //intersect_time
	unsigned long intersect_time = report();

	unsigned long total_time = data_prep_time + 
							   sort_time +
							   pre_sort_time +
							   intersect_time;
	printf("sort\t"
		   "total:%lu\t"
		   "prep:%lu,%f\t"
		   "sort:%lu,%f\t"
		   "presort:%lu,%f\t"
		   "intersect:%lu,%f\n",
		   total_time,
		   data_prep_time,  (double)data_prep_time / (double)total_time,
		   sort_time, (double)sort_time / (double)total_time,
		   pre_sort_time, (double)pre_sort_time / (double)total_time,
		   intersect_time, (double)intersect_time / (double)total_time);

	hipFree(A_starts_d);
	hipFree(A_lens_d);
	hipFree(B_starts_d);
	hipFree(B_ends_d);
	hipFree(R_d);

	return R;
}
//}}}

//{{{ unsigned int count_intersections_i_bsearch_cuda(struct interval *A,
unsigned int count_intersections_i_gm_bsearch_cuda(struct interval *A,
										      unsigned int size_A,
											  struct interval *B,
										      unsigned int size_B,
										      unsigned int size_I)
{
	int block_size = 256;
	dim3 dimBlock(block_size);
	int grid_size = ( size_A + block_size - 1) / (block_size);
	dim3 dimGridSearch( grid_size );
	hipError_t err;

	start(); //data_prep_time

	//{{{ Allocate and move 
	unsigned int *A_starts_h, *A_lens_h, *B_starts_h, *B_ends_h;
	unsigned int *A_starts_d, *A_lens_d, *B_starts_d, *B_ends_d;
	unsigned int *R_d;
	allocate_and_move(A,
					&A_starts_h,
					&A_starts_d,
					&A_lens_h ,
					&A_lens_d,
					size_A,
					B,
					&B_starts_h ,
					&B_starts_d,
					&B_ends_h ,
					&B_ends_d,
					size_B,
					&R_d);
	//}}}
	//
	stop(); //data_prep_time
	unsigned long data_prep_time = report();

	start();//sort_time
	//{{{ Sort B_starts and B_ends
	// Sort B by start
	nvRadixSort::RadixSort radixsortB_starts(size_B, true);
	radixsortB_starts.sort((unsigned int*)B_starts_d, 0, size_B, 32);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_starts: %s.\n", hipGetErrorString( err) );

	// Sort B by end
	nvRadixSort::RadixSort radixsortB_ends(size_B, true);
	radixsortB_ends.sort((unsigned int*)B_ends_d, 0, size_B, 32);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Sort B_ends: %s.\n", hipGetErrorString( err) );
	//}}}
	stop();	//sort_time
	unsigned long sort_time = report();

	start();//index_time
	//{{{ Generate index
	unsigned int *I_starts_d, *I_ends_d;
	hipMalloc((void **)&I_starts_d, (size_I)*sizeof(unsigned int));
	hipMalloc((void **)&I_ends_d, (size_I)*sizeof(unsigned int));

	int index_grid_size = ( size_I + block_size - 1) / (block_size);
	dim3 index_dimGrid( index_grid_size );

	gen_index <<<index_dimGrid, dimBlock>>> ( B_starts_d, size_B, I_starts_d, size_I);
	gen_index <<<index_dimGrid, dimBlock>>> ( B_ends_d, size_B, I_ends_d, size_I);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Count i bsearch: %s.\n", hipGetErrorString( err) );
	//}}}
	stop();	//index_time
	unsigned long index_time = report();

	//{{{ Compute and count intersections
	count_i_gm_bsearch_cuda <<<dimGridSearch, dimBlock >>> (
			A_starts_d, A_lens_d, size_A,
			B_starts_d, B_ends_d, size_B,
			I_starts_d, I_ends_d, size_I,
			R_d,
			1);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Count i bsearch: %s.\n", hipGetErrorString( err) );

	parallel_sum(R_d, block_size, size_A, 1024);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Parallel sum: %s.\n", hipGetErrorString( err) );


	unsigned int R;
	hipMemcpy(&R, R_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Result move: %s.\n", hipGetErrorString( err) );

	//}}}
	stop(); //intersect_time
	unsigned long intersect_time = report();

	unsigned long total_time = data_prep_time + 
							   sort_time +
							   index_time +
							   intersect_time;
	printf("index gm\t"
		   "total:%lu\t"
		   "prep:%lu,%f\t"
		   "sort:%lu,%f\t"
		   "index:%lu,%f\t"
		   "intersect:%lu,%f\n",
		   total_time,
		   data_prep_time,  (double)data_prep_time / (double)total_time,
		   sort_time, (double)sort_time / (double)total_time,
		   index_time, (double)index_time / (double)total_time,
		   intersect_time, (double)intersect_time / (double)total_time);

	hipFree(A_starts_d);
	hipFree(A_lens_d);
	hipFree(B_starts_d);
	hipFree(B_ends_d);
	hipFree(I_starts_d);
	hipFree(I_ends_d);
	hipFree(R_d);

	return R;
}
//}}}

//{{{ __global__ void count_bsearch_cuda (	unsigned int *A_start,
/*
 * @param A_start list of start positions to query, does not need to be sorted
 * @param A_len list of lengths that correspond to A_start
 * @param A_size size of A_start and A_len
 * @param B_start list of sorted start positions to be queried
 * @param B_end list of sorted end positions to be queired 
 * @param B_size size of B_start and B_end
 * @param R number of intersections for each interval in A
 * @param n number of intervals per thread
 */
__global__
void count_bsearch_cuda (	unsigned int *A_start,
							unsigned int *A_len,
							int A_size,
							unsigned int *B_start,
							unsigned int *B_end,
							int B_size,
							unsigned int *R,
							int n)
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	unsigned int i = id;
	unsigned int grid_size = blockDim.x * gridDim.x;

	while ( i < (n * grid_size) ) {

		if (i < A_size) {
			unsigned int start = A_start[i];
			unsigned int end = start + A_len[i];

			int cant_before = bound_binary_search(B_end,
														   B_size,
														   start,
														   -1,
														   B_size);

			int cant_after = bound_binary_search(B_start,
														  B_size,
														  end,
														  -1,
														  B_size);

			while ( end == B_start[cant_after] )
				++cant_after;

			cant_after = A_size - cant_after;	

			R[i] = A_size - cant_before - cant_after;
		}
		i += grid_size;
	}
}
//}}}

//{{{ __global__ void count_i_gm_bsearch_cuda (	unsigned int *A_start,
/*
 * @param A_start list of start positions to query, does not need to be sorted
 * @param A_len list of lengths that correspond to A_start
 * @param A_size size of A_start and A_len
 * @param B_start list of sorted start positions to be queried
 * @param B_end list of sorted end positions to be queired 
 * @param B_size size of B_start and B_end
 * @param R number of intersections for each interval in A
 * @param n number of intervals per thread
 */
__global__
void count_i_gm_bsearch_cuda (	unsigned int *A_start,
							unsigned int *A_len,
							int A_size,
							unsigned int *B_start,
							unsigned int *B_end,
							int B_size,
							unsigned int *I_start,
							unsigned int *I_end,
							int I_size,
							unsigned int *R,
							int n)
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	unsigned int i = id;
	unsigned int grid_size = blockDim.x * gridDim.x;

	while ( i < (n * grid_size) ) {
		if (i < A_size) {
			unsigned int start = A_start[i];
			unsigned int end = start + A_len[i];

			int cant_before = i_binary_search(B_end,
											  B_size,
											  start,
											  I_end,
											  I_size);
	
			int cant_after = i_binary_search(B_start,
											 B_size,
											 end,
											 I_start,
											 I_size);

			while ( end == B_start[cant_after] )
				++cant_after;

			cant_after = A_size - cant_after;	

			R[i] = A_size - cant_before - cant_after;
		}
		i += grid_size;
	}
}
//}}}

//{{{void allocate_and_move( struct interval *A,
void allocate_and_move( struct interval *A,
						unsigned int **A_starts_h,
						unsigned int **A_starts_d,
					   	unsigned int **A_lens_h ,
						unsigned int **A_lens_d,
						unsigned int size_A,

						struct interval *B,
						unsigned int **B_starts_h ,
						unsigned int **B_starts_d,
						unsigned int **B_ends_h ,
						unsigned int **B_ends_d,
						unsigned int size_B,

						unsigned int **R_d)
{

	hipError_t err;
	//{{{ Move intervals to unsigned int arrays
	*A_starts_h = (unsigned int *) malloc( (size_A) * sizeof(unsigned int));
	*A_lens_h = (unsigned int *) malloc( (size_A) * sizeof(unsigned int));

	*B_starts_h = (unsigned int *) malloc( (size_B) * sizeof(unsigned int));
	*B_ends_h = (unsigned int *) malloc( (size_B) * sizeof(unsigned int));

	int i;
	for (i = 0; i < size_B; i++) {
		(*B_starts_h)[i] = B[i].start;
		(*B_ends_h)[i] = B[i].end;
	}

	for (i = 0; i < size_A; i++) {
		(*A_starts_h)[i] = A[i].start;
		(*A_lens_h)[i] = A[i].end - A[i].start;
	}
	//}}}

	//{{{ Move inteval arrays to device
	hipMalloc((void **)A_starts_d, (size_A)*sizeof(unsigned int));
	hipMalloc((void **)A_lens_d, (size_A)*sizeof(unsigned int));
	hipMalloc((void **)B_starts_d, (size_B)*sizeof(unsigned int));
	hipMalloc((void **)B_ends_d, (size_B)*sizeof(unsigned int));

	hipMemcpy(*A_starts_d, *A_starts_h, (size_A) * sizeof(unsigned int), 
			hipMemcpyHostToDevice);
	hipMemcpy(*A_lens_d, *A_lens_h, (size_A) * sizeof(unsigned int),
			hipMemcpyHostToDevice);
	hipMemcpy(*B_starts_d, *B_starts_h, (size_B) * sizeof(unsigned int), 
			hipMemcpyHostToDevice);
	hipMemcpy(*B_ends_d, *B_ends_h, (size_B) * sizeof(unsigned int),
			hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "Interval move: %s.\n", hipGetErrorString( err) );
	//}}}
	
	//{{{ Alocate space for result on device
	hipMalloc((void **)R_d, (size_A)*sizeof(unsigned int));
	unsigned long memup_time = report();

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "R_d malloc: %s.\n", hipGetErrorString( err) );
	//}}}
}
//}}}
